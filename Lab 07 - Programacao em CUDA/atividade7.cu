// ATIVIDADE 7 - PROGRAMACAO EM CUDA
// Ana Julia de Oliveira Bellini - RA 111.774
// Willian Dihanster Gomes de Oliveira - RA 112.269

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T 0.01
#define kappa 0.000045
#define THREADS_PER_BLOCK 512

typedef long long int nint;

__global__ void calcular(double *u, double *prev, double k, double t, double x, int n) {
    nint i =  blockDim.x * blockIdx.x + threadIdx.x;
    if (i > 0 && i < (n - 1))
        u[i] = prev[i] + k * t / (x * x) * (prev[i-1] - 2 * prev[i] + prev[i+1]);
}

__global__ void contorno(double *u, int n) {
    nint i =  blockDim.x * blockIdx.x + threadIdx.x;
    if(i == 1) u[0] = u[n] = 0.0;
}

int main(void) {
  double *tmp, *u, *u_prev;
  double x = dx, t;
  nint i, n, maxloc, size, num_blocos;
  double *d_u, *d_u_prev;

  /* Calculando quantidade de pontos */
  n = tam / dx;
  num_blocos = ceil((n + 1) / THREADS_PER_BLOCK);

  /* Alocando vetores */
  u = (double *) malloc((n + 1) * sizeof(double));
  u_prev = (double *) malloc((n + 1) * sizeof(double));

  size = (n + 1) * sizeof(nint);
  hipMalloc((void **) &d_u, size);
  hipMalloc((void **) &d_u_prev, size);

  printf("Inicio: qtde=%ld, dt=%g, dx=%g, dx²=%g, kappa=%f, const=%f\n",
	 (n+1), dt, dx, dx*dx, kappa, kappa*dt/(dx*dx));
  printf("Iteracoes previstas: %g\n", T/dt);

  x = 0;
  for (i = 0; i < n + 1; i++) {
      if (x <= 0.5)
          u_prev[i] = 200 * x;
      else
          u_prev[i] = 200 * (1.-x);
      x += dx;
  }

  printf("dx=%g, x=%g, x-dx=%g\n", dx, x, x-dx);
  printf("u_prev[0,1]=%g, %g\n",u_prev[0],u_prev[1]);
  printf("u_prev[n-1,n]=%g, %g\n",u_prev[n-1],u_prev[n]);

  hipMemcpy(d_u_prev, u_prev, size, hipMemcpyHostToDevice);

  t = 0.;
  while (t < T) {
    x = dx;

    /* Chama a funcao de que sera realizada no device */
    calcular <<<num_blocos, THREADS_PER_BLOCK>>> (d_u, d_u_prev, kappa, dt, dx, n + 1);

    /* Espera que toda threads terminem os calculos */
    hipDeviceSynchronize();

    /* Agora deve calcular a condicao de contorno */
    contorno <<<num_blocos, THREADS_PER_BLOCK>>> (d_u, n);

    /* Troca entre ponteiros */
    tmp = d_u_prev; d_u_prev = d_u; d_u = tmp;

    t += dt;
  }

  hipMemcpy(u, d_u, size, hipMemcpyDeviceToHost);

  /* Calculando o maior valor e sua localizacao */
  maxloc = 0;
  for (i = 1; i < n + 1; i++)
      if (u[i] > u[maxloc]) maxloc = i;

  printf("Maior valor u[%ld] = %g\n", maxloc, u[maxloc]);

  /* Liberando a memeoria do host */
  free(u);
  free(u_prev);
  u = NULL;
  u_prev = NULL;

  /* Liberando a memeoria do device */
  hipFree(d_u);
  hipFree(d_u_prev);

  return 0;
}
